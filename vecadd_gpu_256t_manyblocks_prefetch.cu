#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>

__global__ void add(int n, float* x, float* y)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride)
    {
        y[i] = x[i] + y[i];
    }
}

int main()
{
    int N = 1 << 20; // 1M elements
    float* x, *y;

    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Prefetch memory to GPU
    int deviceID = 0;
    hipMemPrefetchAsync((void*)x, N * sizeof(float), deviceID);
    hipMemPrefetchAsync((void*)y, N * sizeof(float), deviceID);

    add<<<1, 1>>>(N, x, y);

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
    }

    // Cleanup
    hipFree(x);
    hipFree(y);

    return 0;
}
