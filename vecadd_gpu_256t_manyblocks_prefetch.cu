#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>

__global__ void add(int n, float* x, float* y)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride)
    {
        y[i] = x[i] + y[i];
    }
}

int main()
{
    int N = 1 << 26; // 6M elements
    float* x, *y;

    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Prefetch memory to GPU
    int deviceID = 0;
    hipMemPrefetchAsync((void*)x, N * sizeof(float), deviceID);
    hipMemPrefetchAsync((void*)y, N * sizeof(float), deviceID);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    add<<<blocksPerGrid, threadsPerBlock>>>(N, x, y);

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
    }

    // Cleanup
    hipFree(x);
    hipFree(y);

    return 0;
}
